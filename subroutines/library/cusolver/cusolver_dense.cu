#include <stdio.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

#define CHECK_CUSOLVER(call, msg) { \
    hipsolverStatus_t err = call; \
    if (err != HIPSOLVER_STATUS_SUCCESS) { \
        fprintf(stderr, "%s\n", msg); \
        fprintf(stderr, "Error code: %d\n", err); \
        exit(1); \
    } \
}

int main() {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH), "Failed to create cuSolver handle.");
    CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamDefault));
    CHECK_CUSOLVER(hipsolverSetStream(cusolverH, stream), "Failed to set stream for cuSolver.");

    // example code for solving Ax=b
    const int n = 3; // size of matrix
    const int lda = n;
    float A[lda*n] = {1, 2, 3, 4, 5, 6, 7, 8, 10};
    float b[n] = {1, 2, 3};
    int info = 0; // error message on device
    float *d_A = NULL, *d_b = NULL;
    int *d_info = NULL; // info on device
    int bufferSize = 0;
    float *buffer = NULL;
    int h_info = 0; // error message on host

    // malloc memory on device
    CHECK_CUDA(hipMalloc((void**)&d_A, sizeof(float) * lda * n));
    CHECK_CUDA(hipMalloc((void**)&d_b, sizeof(float) * n));
    CHECK_CUDA(hipMalloc((void**)&d_info, sizeof(int)));

    // copy data from host to device
    CHECK_CUDA(hipMemcpy(d_A, A, sizeof(float) * lda * n, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, b, sizeof(float) * n, hipMemcpyHostToDevice));

    // check buffer size
    CHECK_CUSOLVER(hipsolverDnSgetrf_bufferSize(cusolverH, n, n, d_A, lda, &bufferSize), "Failed to query buffer size.");
    CHECK_CUDA(hipMalloc(&buffer, sizeof(float) * bufferSize));

    // solve linear system
    CHECK_CUSOLVER(hipsolverDnSgetrf(cusolverH, n, n, d_A, lda, buffer, NULL, d_info), "Failed to compute LU decomposition.");
    CHECK_CUSOLVER(hipsolverDnSgetrs(cusolverH, HIPBLAS_OP_N, n, 1, d_A, lda, NULL, d_b, n, d_info), "Failed to solve linear system.");

    // check error
    CHECK_CUDA(hipMemcpy(&h_info, d_info, sizeof(int), hipMemcpyDeviceToHost));
    if (h_info != 0) {
        fprintf(stderr, "Error: LU decomposition failed\n");
        return 1;
    }

    // copy data from device to host
    CHECK_CUDA(hipMemcpy(b, d_b, sizeof(float) * n, hipMemcpyDeviceToHost));

    printf("Solution: \n");
    for (int i = 0; i < n; i++) {
        printf("%f\n", b[i]);
    }

    // free memory
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_b));
    CHECK_CUDA(hipFree(d_info));
    CHECK_CUDA(hipFree(buffer));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverH), "Failed to destroy cuSolver handle.");
    CHECK_CUDA(hipStreamDestroy(stream));

    return 0;
}
