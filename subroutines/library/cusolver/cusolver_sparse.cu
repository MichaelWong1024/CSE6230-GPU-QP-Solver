
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipsolver.h>
#include <hip/hip_runtime_api.h>

int main() {
    hipsolverSpHandle_t cusolverHandle = NULL;
    hipsparseMatDescr_t descrA = NULL;
    int rowsA = 3;
    int colsA = 3;
    int nnzA = 5;

    // CSR representation of matrix A
    int h_csrRowPtrA[4] = {0, 2, 4, 5};
    int h_csrColIndA[5] = {0, 1, 1, 2, 2};
    double h_csrValA[5] = {4.0, -1.0, -1.0, 4.0, -2.0};
    double h_b[3] = {1.0, 2.0, 3.0}; // vector b

    // allocate memory
    int *d_csrRowPtrA, *d_csrColIndA;
    double *d_csrValA, *d_b, *d_x;
    hipMalloc((void **)&d_csrRowPtrA, sizeof(int) * (rowsA + 1));
    hipMalloc((void **)&d_csrColIndA, sizeof(int) * nnzA);
    hipMalloc((void **)&d_csrValA, sizeof(double) * nnzA);
    hipMalloc((void **)&d_b, sizeof(double) * rowsA);
    hipMalloc((void **)&d_x, sizeof(double) * colsA);
    hipMemcpy(d_csrRowPtrA, h_csrRowPtrA, sizeof(int) * (rowsA + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, h_csrColIndA, sizeof(int) * nnzA, hipMemcpyHostToDevice);
    hipMemcpy(d_csrValA, h_csrValA, sizeof(double) * nnzA, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(double) * rowsA, hipMemcpyHostToDevice);

    // create cuSolver and cuSparse handle
    hipsolverSpCreate(&cusolverHandle);
    hipsparseCreateMatDescr(&descrA);

    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    int singularity = 0; // check if matrix is singular

    // solve linear system
    hipsolverSpDcsrlsvqr(cusolverHandle, rowsA, nnzA, descrA,
                        d_csrValA, d_csrRowPtrA, d_csrColIndA, d_b,
                        0.0,
                        0,
                        d_x,
                        &singularity);

    if (singularity >= 0) {
        printf("Matrix is singular at %d\n", singularity);
    } else {
        double h_x[3];
        hipMemcpy(h_x, d_x, sizeof(double) * colsA, hipMemcpyDeviceToHost);
        printf("Solution: \n");
        for (int i = 0; i < colsA; i++) {
            printf("%f\n", h_x[i]);
        }
    }

    // free memory
    hipsolverSpDestroy(cusolverHandle);
    hipsparseDestroyMatDescr(descrA);
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);
    hipFree(d_csrValA);
    hipFree(d_b);
    hipFree(d_x);

    return 0;
}
