#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

#define CHECK_CUBLAS(call, msg) { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "%s\n", msg); \
        fprintf(stderr, "Error code: %d\n", status); \
        exit(1); \
    } \
}

int main() {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    CHECK_CUBLAS(hipblasCreate(&cublasH), "Failed to create cuBLAS handle.");
    CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamDefault));
    CHECK_CUBLAS(hipblasSetStream(cublasH, stream), "Failed to set stream for cuBLAS.");

    // Example cuBLAS operation: matrix-matrix multiplication (C = A * B)
    const int m = 3, k = 3, n = 3;
    float alpha = 1.0f;
    float beta = 0.0f;
    float A[m*k] = {1, 4, 7, 2, 5, 8, 3, 6, 9};
    float B[k*n] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    float C[m*n];
    float *d_A, *d_B, *d_C;

    CHECK_CUDA(hipMalloc((void**)&d_A, sizeof(float) * m * k));
    CHECK_CUDA(hipMalloc((void**)&d_B, sizeof(float) * k * n));
    CHECK_CUDA(hipMalloc((void**)&d_C, sizeof(float) * m * n));
    CHECK_CUDA(hipMemcpy(d_A, A, sizeof(float) * m * k, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, B, sizeof(float) * k * n, hipMemcpyHostToDevice));

    CHECK_CUBLAS(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m), "Failed to perform matrix-matrix multiplication.");

    CHECK_CUDA(hipMemcpy(C, d_C, sizeof(float) * m * n, hipMemcpyDeviceToHost));

    printf("Result of matrix multiplication (C = A * B):\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f ", C[i * n + j]);
        }
        printf("\n");
    }

    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUBLAS(hipblasDestroy(cublasH), "Failed to destroy cuBLAS handle.");
    CHECK_CUDA(hipStreamDestroy(stream));

    return 0;
}
